#include "GSZ_entry.h"
#include "GSZ.h"

/** ************************************************************************
 * @brief GSZ end-to-end compression API for host pointers
 *        Compression is executed in GPU.
 *        Original data is stored as host pointers (in CPU).
 *        Compressed data is stored back as host pointers (in CPU).
 * 
 * @param   oriData         original data (host pointer)
 * @param   cmpBytes        compressed data (host pointer)
 * @param   nbEle           original data size (number of floating point)
 * @param   cmpSize         compressed data size (number of unsigned char)
 * @param   errorBound      user-defined error bound
 * *********************************************************************** */
void GSZ_compress_hostptr(float* oriData, unsigned char* cmpBytes, size_t nbEle, size_t* cmpSize, float errorBound)
{
    // Data blocking.
    int bsize = cmp_tblock_size;
    int gsize = (nbEle + bsize * cmp_chunk - 1) / (bsize * cmp_chunk);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * cmp_chunk;

    // Initializing global memory for GPU compression.
    float* d_oriData;
    unsigned char* d_cmpData;
    unsigned int* d_cmpOffset;
    unsigned int* d_locOffset;
    int* d_flag;
    unsigned int glob_sync;
    hipMalloc((void**)&d_oriData, sizeof(float)*pad_nbEle);
    hipMemcpy(d_oriData, oriData, sizeof(float)*pad_nbEle, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_cmpData, sizeof(float)*pad_nbEle);
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_locOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_locOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // Initializing CUDA Stream.
    hipStream_t stream;
    hipStreamCreate(&stream);

    // GSZ GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    GSZ_compress_kernel_plain<<<gridSize, blockSize, sizeof(unsigned int)*2, stream>>>(d_oriData, d_cmpData, d_cmpOffset, d_locOffset, d_flag, errorBound, nbEle);

    // Obtain compression ratio and move data back to CPU.  
    hipMemcpy(&glob_sync, d_cmpOffset+cmpOffSize-2, sizeof(unsigned int), hipMemcpyDeviceToHost);
    // Yafan@2023 Sep.20: Didn't add the last block info, so the compression is slightly compromised. 
    //                    Temporarilly adding one gsize to solve this. 
    //                    More solutions will be added in the future.
    // Yafan@2023 Oct.21: New update can be found in cuSZp open-source repo.
    *cmpSize = (size_t)glob_sync + pad_nbEle/32 + 2*gsize;
    hipMemcpy(cmpBytes, d_cmpData, *cmpSize*sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Free memory that is used.
    hipFree(d_oriData);
    hipFree(d_cmpData);
    hipFree(d_cmpOffset);
    hipFree(d_locOffset);
    hipFree(d_flag);
    hipStreamDestroy(stream);
}

/** ************************************************************************
 * @brief GSZ end-to-end decompression API for host pointers
 *        Decompression is executed in GPU.
 *        Compressed data is stored as host pointers (in CPU).
 *        Reconstructed data is stored back as host pointers (in CPU).
 *        P.S. Reconstructed data and original data have the same shape.
 * 
 * @param   decData         reconstructed data (host pointer)
 * @param   cmpBytes        compressed data (host pointer)
 * @param   nbEle           reconstructed data size (number of floating point)
 * @param   cmpSize         compressed data size (number of unsigned char)
 * @param   errorBound      user-defined error bound
 * *********************************************************************** */
void GSZ_decompress_hostptr(float* decData, unsigned char* cmpBytes, size_t nbEle, size_t cmpSize, float errorBound)
{
    // Data blocking.
    int bsize = dec_tblock_size;
    int gsize = (nbEle + bsize * dec_chunk - 1) / (bsize * dec_chunk);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * dec_chunk;

    // Initializing global memory for GPU compression.
    float* d_decData;
    unsigned char* d_cmpData;
    unsigned int* d_cmpOffset;
    unsigned int* d_locOffset;
    int* d_flag;
    hipMalloc((void**)&d_decData, sizeof(float)*pad_nbEle);
    hipMemset(d_decData, 0, sizeof(float)*pad_nbEle);
    hipMalloc((void**)&d_cmpData, sizeof(float)*pad_nbEle);
    hipMemcpy(d_cmpData, cmpBytes, sizeof(unsigned char)*cmpSize, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_locOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_locOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // Initializing CUDA Stream.
    hipStream_t stream;
    hipStreamCreate(&stream);

    // GSZ GPU decompression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    GSZ_decompress_kernel_plain<<<gridSize, blockSize, sizeof(unsigned int)*2, stream>>>(d_decData, d_cmpData, d_cmpOffset, d_locOffset, d_flag, errorBound, nbEle);

    // Move data back to CPU.
    hipMemcpy(decData, d_decData, sizeof(float)*pad_nbEle, hipMemcpyDeviceToHost);

    // Free memoy that is used.
    hipFree(d_decData);
    hipFree(d_cmpData);
    hipFree(d_cmpOffset);
    hipFree(d_locOffset);
    hipFree(d_flag);
    hipStreamDestroy(stream);
}

/** ************************************************************************
 * @brief GSZ end-to-end compression API for device pointers
 *        Compression is executed in GPU.
 *        Original data is stored as device pointers (in GPU).
 *        Compressed data is stored back as device pointers (in GPU).
 * 
 * @param   d_oriData       original data (device pointer)
 * @param   d_cmpBytes      compressed data (device pointer)
 * @param   nbEle           original data size (number of floating point)
 * @param   cmpSize         compressed data size (number of unsigned char)
 * @param   errorBound      user-defined error bound
 * @param   stream          CUDA stream for executing compression kernel
 * *********************************************************************** */
void GSZ_compress_deviceptr_plain(float* d_oriData, unsigned char* d_cmpBytes, size_t nbEle, size_t* cmpSize, float errorBound, hipStream_t stream)
{
    // Data blocking.
    int bsize = cmp_tblock_size;
    int gsize = (nbEle + bsize * cmp_chunk - 1) / (bsize * cmp_chunk);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU compression.
    unsigned int* d_cmpOffset;
    unsigned int* d_locOffset;
    int* d_flag;
    unsigned int glob_sync;
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_locOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_locOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // GSZ GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    GSZ_compress_kernel_plain<<<gridSize, blockSize, sizeof(unsigned int)*2, stream>>>(d_oriData, d_cmpBytes, d_cmpOffset, d_locOffset, d_flag, errorBound, nbEle);

    // Obtain compression ratio and move data back to CPU.  
    hipMemcpy(&glob_sync, d_cmpOffset+cmpOffSize-2, sizeof(unsigned int), hipMemcpyDeviceToHost);
    *cmpSize = (size_t)glob_sync + (nbEle+cmp_tblock_size*cmp_chunk-1)/(cmp_tblock_size*cmp_chunk)*(cmp_tblock_size*cmp_chunk)/32;

    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_locOffset);
    hipFree(d_flag);
}

 /** ************************************************************************
 * @brief GSZ end-to-end decompression API for device pointers
 *        Decompression is executed in GPU.
 *        Compressed data is stored as device pointers (in GPU).
 *        Reconstructed data is stored as device pointers (in GPU).
 *        P.S. Reconstructed data and original data have the same shape.
 * 
 * @param   d_decData       reconstructed data (device pointer)
 * @param   d_cmpBytes      compressed data (device pointer)
 * @param   nbEle           reconstructed data size (number of floating point)
 * @param   cmpSize         compressed data size (number of unsigned char)
 * @param   errorBound      user-defined error bound
 * @param   stream          CUDA stream for executing compression kernel
 * *********************************************************************** */
void GSZ_decompress_deviceptr_plain(float* d_decData, unsigned char* d_cmpBytes, size_t nbEle, size_t cmpSize, float errorBound, hipStream_t stream)
{
    // Data blocking.
    int bsize = dec_tblock_size;
    int gsize = (nbEle + bsize * dec_chunk - 1) / (bsize * dec_chunk);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU decompression.
    unsigned int* d_cmpOffset;
    unsigned int* d_locOffset;
    int* d_flag;
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_locOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_locOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // GSZ GPU decompression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    GSZ_decompress_kernel_plain<<<gridSize, blockSize, sizeof(unsigned int)*2, stream>>>(d_decData, d_cmpBytes, d_cmpOffset, d_locOffset, d_flag, errorBound, nbEle);
    
    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_locOffset);
    hipFree(d_flag);
}

/** ************************************************************************
 * @brief GSZ end-to-end compression API for device pointers
 *        Compression is executed in GPU.
 *        Original data is stored as device pointers (in GPU).
 *        Compressed data is stored back as device pointers (in GPU).
 * 
 * @param   d_oriData       original data (device pointer)
 * @param   d_cmpBytes      compressed data (device pointer)
 * @param   nbEle           original data size (number of floating point)
 * @param   cmpSize         compressed data size (number of unsigned char)
 * @param   errorBound      user-defined error bound
 * @param   stream          CUDA stream for executing compression kernel
 * *********************************************************************** */
void GSZ_compress_deviceptr_outlier(float* d_oriData, unsigned char* d_cmpBytes, size_t nbEle, size_t* cmpSize, float errorBound, hipStream_t stream)
{
    // Data blocking.
    int bsize = cmp_tblock_size;
    int gsize = (nbEle + bsize * cmp_chunk - 1) / (bsize * cmp_chunk);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU compression.
    unsigned int* d_cmpOffset;
    unsigned int* d_locOffset;
    int* d_flag;
    unsigned int glob_sync;
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_locOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_locOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // GSZ GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    GSZ_compress_kernel_outlier<<<gridSize, blockSize, sizeof(unsigned int)*2, stream>>>(d_oriData, d_cmpBytes, d_cmpOffset, d_locOffset, d_flag, errorBound, nbEle);

    // Obtain compression ratio and move data back to CPU.  
    hipMemcpy(&glob_sync, d_cmpOffset+cmpOffSize-2, sizeof(unsigned int), hipMemcpyDeviceToHost);
    *cmpSize = (size_t)glob_sync + (nbEle+cmp_tblock_size*cmp_chunk-1)/(cmp_tblock_size*cmp_chunk)*(cmp_tblock_size*cmp_chunk)/32;

    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_locOffset);
    hipFree(d_flag);
}

 /** ************************************************************************
 * @brief GSZ end-to-end decompression API for device pointers
 *        Decompression is executed in GPU.
 *        Compressed data is stored as device pointers (in GPU).
 *        Reconstructed data is stored as device pointers (in GPU).
 *        P.S. Reconstructed data and original data have the same shape.
 * 
 * @param   d_decData       reconstructed data (device pointer)
 * @param   d_cmpBytes      compressed data (device pointer)
 * @param   nbEle           reconstructed data size (number of floating point)
 * @param   cmpSize         compressed data size (number of unsigned char)
 * @param   errorBound      user-defined error bound
 * @param   stream          CUDA stream for executing compression kernel
 * *********************************************************************** */
void GSZ_decompress_deviceptr_outlier(float* d_decData, unsigned char* d_cmpBytes, size_t nbEle, size_t cmpSize, float errorBound, hipStream_t stream)
{
    // Data blocking.
    int bsize = dec_tblock_size;
    int gsize = (nbEle + bsize * dec_chunk - 1) / (bsize * dec_chunk);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU decompression.
    unsigned int* d_cmpOffset;
    unsigned int* d_locOffset;
    int* d_flag;
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_locOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_locOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // GSZ GPU decompression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    GSZ_decompress_kernel_outlier<<<gridSize, blockSize, sizeof(unsigned int)*2, stream>>>(d_decData, d_cmpBytes, d_cmpOffset, d_locOffset, d_flag, errorBound, nbEle);
    
    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_locOffset);
    hipFree(d_flag);
}