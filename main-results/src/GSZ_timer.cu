#include "GSZ_timer.h"

/** ************************************************************************
 * @brief           CUDA event timer for measuring GPU kernel execution.
 * *********************************************************************** */
TimingGPU::TimingGPU() { privateTimingGPU = new PrivateTimingGPU;  }
TimingGPU::~TimingGPU() { }

/** ************************************************************************
 * @brief           Start timer.
 * *********************************************************************** */
void TimingGPU::StartCounter()
{
    hipEventCreate(&((*privateTimingGPU).start));
    hipEventCreate(&((*privateTimingGPU).stop));
    hipEventRecord((*privateTimingGPU).start,0);
}

/** ************************************************************************
 * @brief           Start timer with flags.
 * *********************************************************************** */
void TimingGPU::StartCounterFlags()
{
    int eventflags = hipEventBlockingSync;

    hipEventCreateWithFlags(&((*privateTimingGPU).start),eventflags);
    hipEventCreateWithFlags(&((*privateTimingGPU).stop),eventflags);
    hipEventRecord((*privateTimingGPU).start,0);
}

/** ************************************************************************
 * @brief           End timer, get count in ms.
 * *********************************************************************** */
float TimingGPU::GetCounter()
{
    float time;
    hipEventRecord((*privateTimingGPU).stop, 0);
    hipEventSynchronize((*privateTimingGPU).stop);
    hipEventElapsedTime(&time,(*privateTimingGPU).start,(*privateTimingGPU).stop);
    return time;
}
